#include "hip/hip_runtime.h"
#include <vector>
#include <math.h>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>

#include "strided_batched_gemm.h"

// symbol to be automatically resolved by PyTorch libs
extern THCState *state;

namespace multihead_attn {
namespace self {
namespace matmul1 {

std::vector<torch::Tensor> fwd_cuda(
                               int                  heads,
                               torch::Tensor const& inputs,
                               float                coeff
                             )
{
  
  const int   embed_dim      = inputs.size(2) / 3;
  const int   sequences      = inputs.size(1);
  const int   q_seq_len      = inputs.size(0);
  const int   k_seq_len      = q_seq_len;
  const int   head_dim       = embed_dim / heads;

  const int   attn_batches   = heads * sequences;
  const int   lead_dim       = attn_batches * 3 * head_dim;
  const int   batch_stride   = 3 * head_dim;

  const float beta           = 0.0;
  const float scale          = 1.0 / (sqrt(static_cast<float>(head_dim)) * coeff);

  // There is no reason to use more than one stream as every kernel is
  // sequentially dependent
  hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
  hipStream_t   stream = at::cuda::getCurrentCUDAStream().stream();
  hipblasSetStream(handle, stream);

  auto act_options  = inputs.options().requires_grad(false);
  torch::Tensor outputs   = torch::empty({attn_batches, q_seq_len, k_seq_len},   act_options);

  // Input Linear Results Pointers to Q, K, and V of interviewed activations
  void* inputs_q_ptr   = static_cast<void*>(inputs.data_ptr());
  void* inputs_k_ptr   = static_cast<void*>(static_cast<half*>(inputs.data_ptr()) + head_dim);

  void* outputs_ptr = static_cast<void*>(outputs.data_ptr());

  char a_layout_t{'t'};
  char b_layout_n{'n'};

  BAGUA_CUDABLAS_CHECK(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));

  gemm_switch_fp32accum(     state,
                             a_layout_t,
                             b_layout_n,
                             k_seq_len,
                             q_seq_len,
                             head_dim,
                             scale,
                             static_cast<const half*>(inputs_k_ptr),
                             lead_dim,
                             batch_stride,
                             static_cast<const half*>(inputs_q_ptr),
                             lead_dim,
                             batch_stride,
                             beta,
                             static_cast<half*>(outputs_ptr),
                             k_seq_len,
                             k_seq_len*q_seq_len,
                             attn_batches);


  BAGUA_CUDABLAS_CHECK(hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH));

  return {
      outputs
  };
}

std::vector<torch::Tensor> bwd_cuda(
                               int                  heads,
                               torch::Tensor const& output_grads,
                               torch::Tensor const& inputs,
                               float                coeff
                                   )
{
  const int   embed_dim      = inputs.size(2) / 3;
  const int   sequences      = inputs.size(1);
  const int   q_seq_len      = inputs.size(0);
  const int   k_seq_len      = q_seq_len;
  const int   head_dim       = embed_dim / heads;

  const int   attn_batches   = heads * sequences;
  const int   lead_dim       = attn_batches * 3 * head_dim;
  const int   batch_stride   = 3 * head_dim;

  const float beta           = 0.0;
  const float scale          = 1.0 / (sqrt(static_cast<float>(head_dim)) * coeff);

  // TODO: Streams can be used in Backprop but I haven't added more than one
  // in my first attempt to create the code
  hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
  hipStream_t   stream = at::cuda::getCurrentCUDAStream().stream();
  hipblasSetStream(handle, stream);

  // Output Tensor Allocations
  torch::Tensor inputs_grads   = torch::empty_like(inputs);

  auto inputs_q_ptr = static_cast<half*>(inputs.data_ptr());
  auto inputs_k_ptr = static_cast<half*>(inputs.data_ptr()) + head_dim;

  auto inputs_q_grads_ptr = static_cast<half*>(inputs_grads.data_ptr());
  auto inputs_k_grads_ptr = static_cast<half*>(inputs_grads.data_ptr()) + head_dim;

  char a_layout_n{'n'};
  char b_layout_n{'n'};
  char b_layout_t{'t'};

  BAGUA_CUDABLAS_CHECK(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));

  // Matmul1 Dgrad1
  gemm_switch_fp32accum(     state,
                             a_layout_n,
                             b_layout_n,
                             head_dim,
                             q_seq_len,
                             k_seq_len,
                             scale,
                             inputs_k_ptr,
                             lead_dim,
                             batch_stride,
                             static_cast<half*>(output_grads.data_ptr()),
                             k_seq_len,
                             k_seq_len*q_seq_len,
                             beta,
                             inputs_q_grads_ptr,
                             lead_dim,
                             batch_stride,
                             attn_batches);

  // Matmul1 Dgrad2
  gemm_switch_fp32accum(     state,
                             a_layout_n,
                             b_layout_t,
                             head_dim,
                             k_seq_len,
                             q_seq_len,
                             scale,
                             inputs_q_ptr,
                             lead_dim,
                             batch_stride,
                             static_cast<half*>(output_grads.data_ptr()),
                             k_seq_len,
                             k_seq_len*q_seq_len,
                             beta,
                             inputs_k_grads_ptr,
                             lead_dim,
                             batch_stride,
                             attn_batches);

  BAGUA_CUDABLAS_CHECK(hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH));

  return {
           inputs_grads,
         };
}

} // end namespace matmul1
} // end self
} // end namespace multihead_attn
